#include <chrono>
#include <ctime>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <time.h>
using namespace std;

#define TILE_WIDTH 16

__global__ void matrixMultiply(float *A, float *B, float *C, int numARows,
                               int numAColumns, int numBRows, int numBColumns,
                               int numCRows, int numCColumns) {
  __shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
  __shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];

  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int Row = by * TILE_WIDTH + ty;
  int Col = bx * TILE_WIDTH + tx;
  float Pvalue = 0;
  // ciclo sobre las matrices "shared" M y N  para calcular el
  // elemento P
  for (int m = 0; m < (numAColumns - 1) / TILE_WIDTH + 1; ++m) {
    // Se verifica que tanto como tx como ty, no excedan el tamaño de la
    // matrices, y si lo llegaran a hacer por el tamaño del grid, estos se
    // asignaran como 0
    if (Row < numARows && ((m * TILE_WIDTH) + tx) < numAColumns) {
      ds_M[ty][tx] = A[Row * numAColumns + m * TILE_WIDTH + tx];

    } else {
      ds_M[ty][tx] = 0;
    }
    if ((Col < numBColumns) && ((m * TILE_WIDTH) + ty) < numBRows) {
      ds_N[ty][tx] = B[(m * TILE_WIDTH + ty) * numBColumns + Col];

    } else {
      ds_N[ty][tx] = 0.0;
    }
    __syncthreads();
    for (int k = 0; k < TILE_WIDTH; ++k)
      Pvalue += ds_M[ty][k] * ds_N[k][tx];
    __syncthreads();
  }
  // Solo se guardaran si hilos corresponden a una posicion valida para la
  // matriz resultante
  if (Row < numCRows && Col < numCColumns) {
    C[Row * numCColumns + Col] = Pvalue;
  }
}
void matMultiplyOnHost(float *A, float *B, float *C, int numARows,
                       int numAColumns, int numBRows, int numBColumns,
                       int numCRows, int numCColumns) {
  for (int i = 0; i < numARows; i++) {
    for (int j = 0; j < numBColumns; j++) {
      float result = 0.0;
      for (int k = 0; k < numAColumns; k++) {
        result += A[i * numAColumns + k] * B[k * numBColumns + j];
      }
      C[i * numBColumns + j] = result;
    }
  }
}
void Check(float *m_h, float *m_d, int numCRows, int numCColumns) {
  for (int i = 0; i < numCRows * numCColumns; i++) {
    if (m_h[i] != m_d[i]) {
      cout << "Iqual: False" << endl;
      break;
    }
  }
  cout << "Iqual: True" << endl;
}

int main() {

  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *hostResultC;
  float *deviceA = NULL;
  float *deviceB = NULL;
  float *deviceC = NULL;
  int numArows = 5000;    // number of rows in the matrix A
  int numAColumns = 2500; // number of columns in the matrix A
  int numBrows = 2500;    // number of rows in the matrix B
  int numBColumns = 5000; // number of columns in the matrix B
  int numCrows;
  int numCColumns;

  if (numAColumns == numBrows) {

    numCrows = numArows;
    numCColumns = numBColumns;

    float sizeA = sizeof(float) * numArows * numAColumns;
    float sizeB = sizeof(float) * numBrows * numBColumns;
    float sizeC = sizeof(float) * numCrows * numCColumns;

    // Memoria en host
    // Reservo memoria en el host, la cantidad de columnas x filas x el tamaño
    // de cada dato.
    hostA = (float *)malloc(sizeA);
    hostB = (float *)malloc(sizeB);
    hostC = (float *)malloc(sizeC);
    hostResultC = (float *)malloc(sizeC);

    // Llenamos matrices
    for (int i = 0; i < numArows * numAColumns; i++) {
      hostA[i] = 3;
    }

    for (int i = 0; i < numBrows * numBColumns; i++) {
      hostB[i] = 2;
    }

    // Memoria en device
    std::chrono::time_point<std::chrono::system_clock> start, end;
    std::chrono::duration<double> elapsed_seconds;
    start = std::chrono::system_clock::now();
    // Resevamos memoria en el device, del mismo tamaño que las anteriores
    // matrices.
    hipMalloc((void **)&deviceA, sizeA);
    hipMalloc((void **)&deviceB, sizeB);
    hipMalloc((void **)&deviceC, sizeC);
    end = std::chrono::system_clock::now();
    elapsed_seconds = end - start;
    cout << "Cuda Malloc Time: " << elapsed_seconds.count() << "s\n";

    // Host to Device
    start = std::chrono::system_clock::now();
    // Pasamos la informacion que posee las matrices que estan en el host al
    // device
    hipMemcpy(deviceA, hostA, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB, sizeB, hipMemcpyHostToDevice);
    end = std::chrono::system_clock::now();
    elapsed_seconds = end - start;

    cout << "Cuda Memcpy Host to Device Time: " << elapsed_seconds.count()
         << "s\n";

    start = std::chrono::system_clock::now();
    // Definimos tamaño del Grid y del bloque
    // Donde si tenemos una matriz de MxN, N sera la cantidad de columnas en el
    // grid y M la cantidad de filas en el grid..
    // El tamaño del bloque es Tile_width x Tile_width

    dim3 dimGrid((numCColumns - 1) / TILE_WIDTH + 1,
                 (numCrows - 1) / TILE_WIDTH + 1, 1);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    end = std::chrono::system_clock::now();
    elapsed_seconds = end - start;
    cout << "Dims Time: " << elapsed_seconds.count() << "s\n";

    // Multiplicacion de matrices utilizando tiles en device
    start = std::chrono::system_clock::now();
    // Hago a la función donde le envío las matrices y sus respectivo datos
    matrixMultiply<<<dimGrid, dimBlock>>>(deviceA, deviceB, deviceC, numArows,
                                          numAColumns, numBrows, numBColumns,
                                          numCrows, numCColumns);
    end = std::chrono::system_clock::now();
    elapsed_seconds = end - start;
    cout << "Multplication Device Time: " << elapsed_seconds.count() << "s\n";

    // Device to Host
    start = std::chrono::system_clock::now();

    hipMemcpy(hostC, deviceC, sizeC, hipMemcpyDeviceToHost);
    end = std::chrono::system_clock::now();
    elapsed_seconds = end - start;
    cout << "Cuda Memcmpy Device to Host Time: " << elapsed_seconds.count()
         << "s\n";
    // Multiplication Host
    start = std::chrono::system_clock::now();

    matMultiplyOnHost(hostA, hostB, hostResultC, numArows, numAColumns,
                      numBrows, numBColumns, numCrows, numCColumns);
    end = std::chrono::system_clock::now();
    elapsed_seconds = end - start;
    cout << "Matrix Multiplication Host Time: " << elapsed_seconds.count()
         << "s\n";
    Check(hostC, hostResultC, numCrows, numCColumns);

  } else {
    cout << "Las matrices no se pueden multiplicar " << endl;
  }
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);

  free(hostA);
  free(hostB);
  free(hostC);
  return 0;
}
